
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

void check_error(int err)
{
    if (err != hipSuccess)
    {
        std::cerr << "Error: " << hipGetErrorString((hipError_t)err) << std::endl;
        exit(1);
    }
}

__global__ void matmul_elem(int n, float *a, float *b, float *c)
{
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (column < n && row < n)
    {
        float dot_product = 0;
        for (int i = 0; i < n; i++)
        {
            dot_product += a[row * n + i] * b[i * n + column];
        }
        c[row * n + column] = dot_product;
    }
}

int mult()
{
    float h_A[3][3] = {{1, 2, 3}, {4, 5, 6}, {7, 8, 9}};
    float h_B[3][3] = {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}};
    float h_C[3][3] = {0};

    float *d_A, *d_B, *d_C;
    int size = 3 * 3 * sizeof(float);

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(3, 3);
    dim3 blocksPerGrid(1, 1);
    matmul_elem<<<blocksPerGrid, threadsPerBlock>>>(3, d_A, d_B, d_C);
    check_error(hipGetLastError());

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 3; j++)
        {
            printf("%f ", h_C[i][j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

__global__ void broadcast_elem(int n, float *a, float *b, float *c, float *d)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x < n && y < n && z < n) {
        int index = x * n * n + y * n + z;
        d[index] = a[index] + b[y * n + z] + c[z];
    }
}

int broadcast()
{
    const int n = 3;
    float h_A[n][n][n] = {
        {{1, 1, 1}, {1, 1, 1}, {1, 1, 1}},
        {{1, 1, 1}, {1, 1, 1}, {1, 1, 1}},
        {{1, 1, 1}, {1, 1, 1}, {1, 1, 1}},
    };
    float h_B[n][n] = {{2, 2, 2}, {2, 2, 2}, {2, 2, 2}};
    float h_C[n] = {3, 3, 3};
    float h_D[n][n][n] = {0};

    float *d_A, *d_B, *d_C, *d_D;

    hipMalloc((void **)&d_A, n * n * n * sizeof(float));
    hipMalloc((void **)&d_B, n * n * sizeof(float));
    hipMalloc((void **)&d_C, n * sizeof(float));
    hipMalloc((void **)&d_D, n * n * n * sizeof(float));

    int err;
    err = hipMemcpy(d_A, h_A, n * n * n * sizeof(float), hipMemcpyHostToDevice);
    check_error(err);
    err = hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);
    check_error(err);
    err = hipMemcpy(d_C, h_C, n * sizeof(float), hipMemcpyHostToDevice);
    check_error(err);

    dim3 threadsPerBlock(n, n, n);
    dim3 blocksPerGrid(1, 1, 1);
    broadcast_elem<<<blocksPerGrid, threadsPerBlock>>>(3, d_A, d_B, d_C, d_D);
    check_error(hipGetLastError());

    hipMemcpy(h_D, d_D, sizeof(h_D), hipMemcpyDeviceToHost);

    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            for (int k = 0; k < 3; k++) {
                printf("%f ", h_D[i][j][k]);
            }
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);

    return 0;
}

int main()
{
    // mult();
    broadcast();
    return 0;
}