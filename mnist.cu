
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <fstream>

#include "mnist.h"
#include "kernels.h"
#include "utils.h"
#include "dataset.h"

void NN::init_linear(float *w, float *b, int width, int height, int blockSize) {
    auto [dimGrid, dimBlock] = get_grid2d(width, height, blockSize);
    init_rand<<<dimGrid, dimBlock>>>(width, height, w);

    std::tie(dimGrid, dimBlock) = get_grid1d(height, blockSize);
    init_rand<<<dimGrid, dimBlock>>>(1, height, b);
}

void NN::init() {
    hipMalloc(&w1, size1 * input_size * sizeof(float));
    hipMalloc(&b1, size1 * sizeof(float));
    hipMalloc(&d_l1, size1*batch_size*sizeof(float));
    this->init_linear(w1, b1, size1, input_size, threadsPerBlock);

    hipMalloc(&w2, size2 * size1 * sizeof(float));
    hipMalloc(&b2, size2 * sizeof(float));
    hipMalloc(&d_l2, size2*batch_size*sizeof(float));
    init_linear(w2, b2, size2, size1, threadsPerBlock);

    hipMalloc(&w3, size3 * size2 * sizeof(float));
    hipMalloc(&b3, size3 * sizeof(float));
    hipMalloc(&d_l3, size3*batch_size*sizeof(float));
    init_linear(w3, b3, size3, size2, threadsPerBlock);

    hipMalloc(&x1, batch_size * size1 * sizeof(float));
    hipMalloc(&a1, batch_size * size1 * sizeof(float));

    hipMalloc(&x2, batch_size * size2 * sizeof(float));
    hipMalloc(&a2, batch_size * size2 * sizeof(float));

    hipMalloc(&x3, batch_size * size3 * sizeof(float));
    hipMalloc(&logits, batch_size * size3 * sizeof(float));

    hipMalloc(&d_input, batch_size * input_size * sizeof(float));
    hipMalloc(&d_y, batch_size * label_size * sizeof(float));
    hipMalloc(&d_loss, batch_size * sizeof(float));

    gpu_check();
}

void NN::forward(float *h_input, float *h_y, bool do_loss) {
    hipMemcpy(d_input, h_input, 
        batch_size * input_size * sizeof(float), 
        hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y,
        batch_size * label_size * sizeof(float), 
        hipMemcpyHostToDevice);
    gpu_check();

    auto [dimGrid, dimBlock] = get_grid2d(size1, batch_size, threadsPerBlock);
    linear<<<dimGrid, dimBlock>>>(batch_size, input_size, size1, d_input, w1, b1, x1);
    relu<<<dimGrid, dimBlock>>>(size1, batch_size, x1, a1);

    std::tie(dimGrid, dimBlock) = get_grid2d(size2, batch_size, threadsPerBlock);
    linear<<<dimGrid, dimBlock>>>(batch_size, size1, size2, a1, w2, b2, x2);
    relu<<<dimGrid, dimBlock>>>(size2, batch_size, x2, a2);

    std::tie(dimGrid, dimBlock) = get_grid2d(size3, batch_size, threadsPerBlock);
    linear<<<dimGrid, dimBlock>>>(batch_size, size2, size3, a2, w3, b3, x3);
    softmax<<<dimGrid, dimBlock>>>(size3, batch_size, x3, logits);

    if (do_loss) {
        std::tie(dimGrid, dimBlock) = get_grid1d(size3, threadsPerBlock);
        cross_entropy<<<dimGrid, dimBlock>>>(size3, batch_size, logits, d_y, d_loss);
    }

    gpu_check();
}

void NN::backward() {
    auto [dimGrid, dimBlock] = get_grid2d(size3, batch_size, threadsPerBlock);
    cross_entropy_backward<<<dimGrid, dimBlock>>>(size3, batch_size, logits, d_y, d_l3);

    std::tie(dimGrid, dimBlock) = get_grid2d(size2, batch_size, threadsPerBlock);
    linear_backward<<<dimGrid, dimBlock>>>(batch_size, size3, size2, w3, b3, d_l3, d_l2);
    relu_backward<<<dimGrid, dimBlock>>>(size2, batch_size, a2, d_l2, d_l2);

    std::tie(dimGrid, dimBlock) = get_grid2d(size1, batch_size, threadsPerBlock);
    linear_backward<<<dimGrid, dimBlock>>>(batch_size, size2, size1, w2, b2, d_l2, d_l1);
    relu_backward<<<dimGrid, dimBlock>>>(size1, batch_size, a1, d_l1, d_l1);
    gpu_check();
}

void NN::update() {
    auto [dimGrid, dimBlock] = get_grid2d(size3, size2, threadsPerBlock);
    linear_update<<<dimGrid, dimBlock>>>(size3, size2, batch_size, lr, w3, b3, a2, d_l3);

    std::tie(dimGrid, dimBlock) = get_grid2d(size2, size1, threadsPerBlock);
    linear_update<<<dimGrid, dimBlock>>>(size2, size1, batch_size, lr, w2, b2, a1, d_l2);

    std::tie(dimGrid, dimBlock) = get_grid2d(size1, input_size, threadsPerBlock);
    linear_update<<<dimGrid, dimBlock>>>(size1, input_size, batch_size, lr, w1, b1, d_input, d_l1);            
    gpu_check();
}

void NN::train(int length, float *h_input, float *h_y, int epochs) {
    float h_loss[batch_size] = { 0 };
    float h_logits[size3 * batch_size] = { 0 };

    for (int epoch = 0; epoch < epochs; epoch++) {
        auto start_time = std::chrono::high_resolution_clock::now();
        int total = 0;
        int correct = 0;
        float cum_loss = 0;

        for (int batch = 0; batch < length / batch_size; batch++) {
            total += batch_size;

            this->forward(
                h_input + batch * batch_size * input_size, 
                h_y + batch * batch_size * label_size, 
                true        // Request loss.
            );
            // Computes loss.
            hipMemcpy(&h_loss, d_loss, batch_size * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&h_logits, logits, size3 * batch_size * sizeof(float), hipMemcpyDeviceToHost);

            for (int i = 0; i < batch_size; i++) {
                int offset = batch * batch_size * label_size + i * label_size;
                int predicted_class = argmax(h_logits + i * label_size, label_size);
                int true_class = argmax(h_y + offset, label_size);
            
                if (predicted_class == true_class) {
                    correct++;
                }
                cum_loss += h_loss[i];
            }
            
            this->backward();
            this->update();
        }

        auto stop_time = std::chrono::high_resolution_clock::now();
        float epoch_time = std::chrono::duration_cast<std::chrono::duration<float, std::milli>>(stop_time - start_time).count();
        std::cout << "Epoch " << epoch << ": " 
            << " took "<< epoch_time << "ms,"
            << " accuracy: " << (float) correct / total 
            << ", loss: " << cum_loss << std::endl;
    }
}

void nn_main() {
    const int train_length = 60000;
    const int test_length = 10000;

    NN nn;
    nn.init();

    float *mnist_train_x = new float[nn.input_size * train_length];
    float *mnist_train_y = new float[nn.label_size * train_length];


    std::ifstream fin("/home/anselm/datasets/mnist/mnist_train.csv");
    if (!fin.is_open()) {
        std::cerr << "Error: Could not open the file mnist_train.csv" << std::endl;
        return;
    }
    read_mnist(fin, 0, train_length, mnist_train_x, mnist_train_y);
    fin.close();

    nn.train(60000, mnist_train_x, mnist_train_y, 60);
}


void test_main() {
    test_linear();
    test_relu();
    test_softmax();
    test_cross_entropy();
}

int main() {
    nn_main();
}